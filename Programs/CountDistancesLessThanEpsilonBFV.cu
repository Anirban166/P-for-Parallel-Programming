#include "hip/hip_runtime.h"
// Author: Anirban166/Anirban Chetia
// E-mail: bloodraven166@gmail.com/ac4743@nau.edu

#include <omp.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <iostream>
#include <complex.h>

// #define N 100000
#define SEED 72
#define BLOCKSIZE 1024

using namespace std; 
struct point { double x, y; };

void warmUpGPU();
__global__ void kernelFunction(struct point * pointData, unsigned long long int * countDistancesLessThanEpsilon, double epsilon);

int main(int argc, char *argv[])
{
  warmUpGPU();	

  // Take epsilon as command line input:
  if (argc != 2)
  { printf("\nIncorrect number of input parameters. Please input an epsilon distance.\n");
	  return 0;
  } 
  double epsilon = atof(argv[1]);
  
  // Initialize point values between [0-1] for x and y:  
  struct point * pointData;
  pointData = (struct point *)malloc(sizeof(struct point) * N); 
  // Display memory required for pointData in mebibytes:
  printf("\nSize of the points data set (MiB): %f", (2.0 * sizeof(double) * N * 1.0) / (1024.0 * 1024.0)); 
  // In gibibytes: (N * 1.0 * sizeof(struct point)/(1024.0 * 1024.0 * 1024.0)) 
  // Seed the rng:
  srand(SEED);
  for (unsigned int i = 0; i < N; i++)
  {
    pointData[i].x = 1000.0 * ((double)(rand()) / RAND_MAX); 
    pointData[i].y = 1000.0 * ((double)(rand()) / RAND_MAX);	 
  }

  /*---
   GPU	
  ---*/

  double tstart = omp_get_wtime();
  hipError_t errCode = hipSuccess;
  if(errCode != hipSuccess)
  {  
    cout << "\nLast error: " << errCode << endl; 
  }

  struct point *dev_pointData;
  unsigned long long int *countDistancesLessThanEpsilon;
  unsigned long long int *dev_countDistancesLessThanEpsilon; 
  countDistancesLessThanEpsilon = (unsigned long long int *)malloc(sizeof(unsigned long long int)); 
  dev_countDistancesLessThanEpsilon = (unsigned long long int *)malloc(sizeof(unsigned long long int));
  *countDistancesLessThanEpsilon = 0;

  // Allocate the entire point data set (pointData) on the device:
  errCode = hipMalloc((struct point**)&dev_pointData, sizeof(struct point) * N);	
  if(errCode != hipSuccess) 
  {
	cout << "\nError: pointData error with code " << errCode << endl; 
  }
  // Allocate the counter variable (countDistancesLessThanEpsilon) on the device:
  errCode = hipMalloc((unsigned long long int**)&dev_countDistancesLessThanEpsilon, sizeof(unsigned long long int));	
  if(errCode != hipSuccess) 
  {
	cout << "\nError: countDistancesLessThanEpsilon error with code " << errCode << endl; 
  }	

  // Copy the point data set to device:
  errCode = hipMemcpy(dev_pointData, pointData, sizeof(struct point) * N, hipMemcpyHostToDevice);
  if(errCode != hipSuccess) 
  {
	cout << "\nError: dev_pointData Memcpy error with code " << errCode << endl; 
  }	
  // Copy the counter variable to device:
  errCode = hipMemcpy(dev_countDistancesLessThanEpsilon, countDistancesLessThanEpsilon, sizeof(unsigned long long int), hipMemcpyHostToDevice);
  if(errCode != hipSuccess) 
  {
	cout << "\nError: countDistancesLessThanEpsilon Memcpy error with code " << errCode << endl; 
  }

  // Calculate block count:
  const unsigned int totalBlocks = ceil(N * 1.0 / 1024.0);
  printf("\nTotal number of blocks (GPU): %d", totalBlocks);

  // Execute kernel:
  double tstartkernel = omp_get_wtime();
  kernelFunction<<<totalBlocks, BLOCKSIZE>>>(dev_pointData, dev_countDistancesLessThanEpsilon, epsilon);
	
  hipDeviceSynchronize();
  double tendkernel = omp_get_wtime();
  printf("\nExecution time for only the GPU kernel (in seconds): %f", tendkernel - tstartkernel);

  if(errCode != hipSuccess)
  {
	cout << "\nError after kernel launch! " << errCode << endl;
  }

  // Copy counter from device (GPU) back to the host (CPU):
  errCode = hipMemcpy(countDistancesLessThanEpsilon, dev_countDistancesLessThanEpsilon, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
  if(errCode != hipSuccess) 
  {
	cout << "\nError: getting result form GPU error with code " << errCode << endl; 
  }

  printf("\nTotal number of distances between points that are within a search radius (epsilon) of %.1f, as computed by the GPU: %llu", epsilon, *countDistancesLessThanEpsilon);
  double tend = omp_get_wtime();
  printf("\nTotal time taken by the GPU (in seconds): %f", tend - tstart);
  
  free(pointData);
  return 0;
}

__global__ void kernelFunction(struct point * pointData, unsigned long long int * countDistancesLessThanEpsilon, double epsilon) 
{
  unsigned int tid = threadIdx.x + (blockIdx.x * blockDim.x); 
  double distance;	
  if (tid >= N)
  {
	  return;
  }
  for(int i = 0; i < N; ++i)
  { 
    distance = sqrt(pow((pointData[tid].x - pointData[i].x), 2) + pow((pointData[tid].y - pointData[i].y), 2));
    if(distance < epsilon)
      atomicAdd(countDistancesLessThanEpsilon, int(1));
  }
  return;
}

__global__ void warmup(unsigned int * tmp) 
{
  if (threadIdx.x == 0)
  *tmp = 69;
  return;
}

void warmUpGPU()
{
  unsigned int * dev_tmp;
  unsigned int * tmp;
  tmp = (unsigned int*)malloc(sizeof(unsigned int));
  *tmp = 0;
  hipError_t errCode = hipSuccess;
  errCode = hipMalloc((unsigned int**)&dev_tmp, sizeof(unsigned int));	
  if(errCode != hipSuccess)
  {
	  cout << "\nError: dev_tmp error with code " << errCode << endl; 
  }
  warmup<<<1, BLOCKSIZE/4>>>(dev_tmp);
  // Copy data from device to host: 
  errCode = hipMemcpy(tmp, dev_tmp, sizeof(unsigned int), hipMemcpyDeviceToHost);
  if(errCode != hipSuccess) 
  {
    cout << "\nError: getting tmp result form GPU error with code " << errCode << endl; 
  }
  // printf("\ntmp (changed to 69 on GPU): %d\n", *tmp);
  hipFree(dev_tmp);
  return;
}